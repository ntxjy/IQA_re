#include "hip/hip_runtime.h"
#include <numeric>
#include <random>

#include <cub/block/block_scan.cuh>
#include <cub/util_ptx.cuh>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "utils/cuda_utils.h"
#include "scan/block_scan.cuh"
#include "scan/commons.h"



template <typename T>
struct ScanOp
{
    __device__ __forceinline__ T operator()(const T & a, const T & b) = delete;
};


template <>
struct ScanOp<float>
{
    __device__ __forceinline__ float operator()(const float & a, const float & b)
    {
        return a + b;
    }
};


template <>
struct ScanOp<float2>
{
    __device__ __forceinline__ float2 operator()(const float2 & a, const float2 & b)
    {
        return {a.x + b.x, a.y + b.y};
    }
};


template <int kBlockX, int kBlockY, int kBlockZ, int kSegLen, typename T>
__global__ void scan(const T * __restrict__ src,
                     T * __restrict__ dst,
                     T * __restrict__ horiAgg,
                     T * __restrict__ vertAgg)
{
    constexpr int kWarpThreads = 32;

    using Scan = mamband::SegBlockScan<T, kSegLen, kBlockX, mamband::BLOCK_SCAN_WARP_SCANS, kBlockY, kBlockZ>;
    __shared__ typename Scan::TempStorage tempStorage;
    Scan scan(tempStorage);

    ScanOp<T> scanOp;

    int tid = threadIdx.y * kBlockX + threadIdx.x;

    int linearWarpId = tid / kWarpThreads;
    int warpIdx = linearWarpId % (kBlockX / kSegLen);
    int warpIdy = linearWarpId / (kBlockX / kSegLen);

    int linearLaneId = hipcub::LaneId();
    int laneIdx = linearLaneId % kSegLen;
    int laneIdy = linearLaneId / kSegLen;

    int gx = warpIdx * kSegLen + laneIdx;
    int gy = warpIdy * (kWarpThreads / kSegLen) + laneIdy;
    int gi = gy * kBlockX + gx;

    T input = src[gi];

    scan.ExclusiveScan(input, input, scanOp, mamband::kHorizontal);
    scan.ExclusiveScan(input, input, scanOp, mamband::kVertical);

    dst[gi] = input;
}


int main()
{
    constexpr dim3 kBlock(16, 16);
    constexpr bool kRandInput = false;

    std::vector<float> matBuf(kBlock.x * kBlock.y, 1.0f);

    if constexpr (kRandInput)
    {
        auto seed = std::random_device()();
        auto e = std::default_random_engine(seed);
        auto d = std::normal_distribution<float>(1.0f, 4.0f);
        auto g = [&d, &e]()
        {
            return d(e);
        };
        std::generate(matBuf.begin(), matBuf.end(), g);
    }

    thrust::host_vector<float> hostScanSrc = matBuf;

    thrust::device_vector<float> devScanSrc = hostScanSrc;
    thrust::device_vector<float> devScanDst(kBlock.x * kBlock.y, 0.0f);
    thrust::device_vector<float> devHoriAgg(kBlock.x * kBlock.y, 0.0f);
    thrust::device_vector<float> devVertAgg(kBlock.x * kBlock.y, 0.0f);

    scan<kBlock.x, kBlock.y, kBlock.z, kBlock.y, float><<<1, kBlock>>>(
            thrust::raw_pointer_cast(devScanSrc.data()),
            thrust::raw_pointer_cast(devScanDst.data()),
            thrust::raw_pointer_cast(devHoriAgg.data()),
            thrust::raw_pointer_cast(devVertAgg.data())
    );
    CUDA_CHECK_LAST_ERROR();
    CUDA_CHECK(hipDeviceSynchronize());

    thrust::host_vector<float> hostScanDst = devScanDst;
    thrust::host_vector<float> hostHoriAgg = devHoriAgg;
    thrust::host_vector<float> hostVertAgg = devVertAgg;

    auto mat = [kBlock, &matBuf](int i, int j) mutable -> float &
    {
        return matBuf[i * kBlock.x + j];
    };

    auto hRes = [kBlock, &hostScanDst](int i, int j) mutable -> float &
    {
        return hostScanDst[i * kBlock.x + j];
    };

    for (int j = 1; j < kBlock.x; ++j)
    {
        mat(0, j) += mat(0, j - 1);
    }

    for (int i = 1; i < kBlock.y; ++i)
    {
        mat(i, 0) += mat(i - 1, 0);
    }

    for (int i = 1; i < kBlock.y; ++i)
    {
        for (int j = 1; j < kBlock.x; ++j)
        {
            mat(i, j) += mat(i, j - 1) + mat(i - 1, j) - mat(i - 1, j - 1);
        }
    }

    printf("cpu\n");
    for (int i = 0; i < kBlock.y; ++i)
    {
        for (int j = 0; j < kBlock.x; ++j)
        {
            printf("%10.2f ", mat(i, j));
        }
        printf("\n");
    }
    printf("\n");

    printf("scan\n");
    for (int i = 0; i < kBlock.y; ++i)
    {
        for (int j = 0; j < kBlock.x; ++j)
        {
            printf("%10.2f ", hRes(i, j));
        }
        printf("\n");
    }
    printf("\n");

    return 0;
}
