/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/

#include <vector>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/extension.h>


#include "scan/commons.h"
#include "selective_scan/global.cuh"
#include "selective_scan/selective_scan.cuh"


#define CHECK_SHAPE(x, ...) TORCH_CHECK(x.sizes() == torch::IntArrayRef({__VA_ARGS__}), #x " must have shape (" #__VA_ARGS__ ")")


#define DISPATCH_ITYPE_FLOAT_AND_HALF_AND_BF16(ITYPE, NAME, ...)                    \
    if (ITYPE == at::ScalarType::Half) {                                            \
        using input_t = at::Half;                                                   \
        __VA_ARGS__();                                                              \
    } else if (ITYPE == at::ScalarType::BFloat16) {                                 \
        using input_t = at::BFloat16;                                               \
        __VA_ARGS__();                                                              \
    } else if (ITYPE == at::ScalarType::Float)  {                                   \
        using input_t = float;                                                      \
        __VA_ARGS__();                                                              \
    } else {                                                                        \
        AT_ERROR(#NAME, " not implemented for input type '", toString(ITYPE), "'"); \
    }


#define DISPATCH_ITYPE_FLOAT_AND_HALF(ITYPE, NAME, ...)                             \
    if (ITYPE == at::ScalarType::Half) {                                            \
        using input_t = at::Half;                                                   \
        __VA_ARGS__();                                                              \
    } else if (ITYPE == at::ScalarType::Float)  {                                   \
        using input_t = float;                                                      \
        __VA_ARGS__();                                                              \
    } else {                                                                        \
        AT_ERROR(#NAME, " not implemented for input type '", toString(ITYPE), "'"); \
    }



#define DISPATCH_WTYPE_FLOAT_AND_HALF_AND_BF16(WTYPE, NAME, ...)                     \
    if (WTYPE == at::ScalarType::Half) {                                             \
        using weight_t = at::Half;                                                   \
        __VA_ARGS__();                                                               \
    } else if (WTYPE == at::ScalarType::BFloat16) {                                  \
        using weight_t = at::BFloat16;                                               \
        __VA_ARGS__();                                                               \
    } else if (WTYPE == at::ScalarType::Float)  {                                    \
        using weight_t = float;                                                      \
        __VA_ARGS__();                                                               \
    } else {                                                                         \
        AT_ERROR(#NAME, " not implemented for weight type '", toString(WTYPE), "'"); \
    }


#define DISPATCH_WTYPE_FLOAT_AND_COMPLEX(WTYPE, NAME, ...)                           \
    if (WTYPE == at::ScalarType::Float) {                                            \
       using weight_t = float;                                                       \
        __VA_ARGS__();                                                               \
    } else if (WTYPE == at::ScalarType::ComplexFloat) {                              \
        using weight_t = c10::complex<float>;                                        \
        __VA_ARGS__();                                                               \
    } else {                                                                         \
        AT_ERROR(#NAME, " not implemented for weight type '", toString(WTYPE), "'"); \
    }


template <typename input_t, typename weight_t, typename output_t>
void selective_scan_bwd_cuda(SSMParamsBwd & params, hipStream_t stream);


void set_ssm_params_fwd(SSMParamsBase & params,
                        // sizes
                        const size_t batch,
                        const size_t dim,
                        const size_t height,
                        const size_t width,
                        const size_t dstate,
                        const size_t n_groups,
                        const size_t maxDimPerBlock,

                        const bool is_variable_B,
                        const bool is_variable_C,
                        // device pointers
                        const at::Tensor u,
                        const at::Tensor delta,
                        const at::Tensor A,
                        const at::Tensor B,
                        const at::Tensor C,
                        const at::Tensor out,
                        const at::Tensor z,
                        const at::Tensor out_z,
                        void * D_ptr,
                        void * delta_bias_ptr,
                        void * x_ptr,
                        bool has_z,
                        bool delta_softplus);


void set_ssm_params_bwd(SSMParamsBwd & params,
                        // sizes
                        const size_t batch,
                        const size_t dim,
                        const size_t height,
                        const size_t width,
                        const size_t dstate,
                        const size_t n_groups,
                        const size_t maxDimPerBlock,

                        const bool is_variable_B,
                        const bool is_variable_C,
                        // device pointers
                        const at::Tensor u,
                        const at::Tensor delta,
                        const at::Tensor A,
                        const at::Tensor B,
                        const at::Tensor C,
                        const at::Tensor z,
                        const at::Tensor out,
                        const at::Tensor out_z,
                        void * D_ptr,
                        void * delta_bias_ptr,
                        void * x_ptr,
                        const at::Tensor dout,
                        const at::Tensor du,
                        const at::Tensor ddelta,
                        const at::Tensor dA,
                        const at::Tensor dB,
                        const at::Tensor dC,
                        const at::Tensor dz,
                        void * dD_ptr,
                        void * ddelta_bias_ptr,
                        bool has_z,
                        bool delta_softplus,
                        bool recompute_out_z,
                        void * rev_shift_tmp_ptr)
{
    // Pass in "dout" instead of "out", we're not gonna use "out" unless we have z
    set_ssm_params_fwd(params,
                       batch,
                       dim,
                       height,
                       width,
                       dstate,
                       n_groups,
                       maxDimPerBlock,
                       is_variable_B,
                       is_variable_C,
                       u,
                       delta,
                       A,
                       B,
                       C,
                       // If not recompute_out_z, pass dout instead of out_z.
                       // This won't be used by the bwd kernel
                       has_z ? out : dout,
                       has_z ? z : dout,
                       recompute_out_z ? out_z : dout,
                       D_ptr,
                       delta_bias_ptr,
                       x_ptr,
                       has_z,
                       delta_softplus);

    if (!recompute_out_z)
    {
        params.out_z_ptr = nullptr;
    }

    // Set the pointers and strides.
    params.dout_ptr = dout.data_ptr();
    params.du_ptr = du.data_ptr();
    params.dA_ptr = dA.data_ptr();
    params.dB_ptr = dB.data_ptr();
    params.dC_ptr = dC.data_ptr();
    params.dD_ptr = dD_ptr;
    params.ddelta_ptr = ddelta.data_ptr();
    params.ddelta_bias_ptr = ddelta_bias_ptr;
    params.dz_ptr = has_z ? dz.data_ptr() : nullptr;

    // All stride are in elements, not bytes.
    params.dout_batch_stride = dout.stride(0);
    params.dout_d_stride = dout.stride(1);
    params.dA_d_stride = dA.stride(0);
    params.dA_dstate_stride = dA.stride(1);

    if (!is_variable_B)
    {
        params.dB_d_stride = dB.stride(0);
    }
    else
    {
        params.dB_batch_stride = dB.stride(0);
        params.dB_group_stride = dB.stride(1);
    }

    params.dB_dstate_stride = !is_variable_B ? dB.stride(1) : dB.stride(2);

    if (!is_variable_C)
    {
        params.dC_d_stride = dC.stride(0);
    }
    else
    {
        params.dC_batch_stride = dC.stride(0);
        params.dC_group_stride = dC.stride(1);
    }

    params.dC_dstate_stride = !is_variable_C ? dC.stride(1) : dC.stride(2);
    params.du_batch_stride = du.stride(0);
    params.du_d_stride = du.stride(1);
    params.ddelta_batch_stride = ddelta.stride(0);
    params.ddelta_d_stride = ddelta.stride(1);

    if (has_z)
    {
        params.dz_batch_stride = dz.stride(0);
        params.dz_d_stride = dz.stride(1);
    }

    params.rev_shift_tmp_ptr = rev_shift_tmp_ptr;
}


std::vector<at::Tensor>
selective_scan_bwd(const at::Tensor & u,
                   const at::Tensor & delta,
                   const at::Tensor & A,
                   const at::Tensor & B,
                   const at::Tensor & C,
                   const c10::optional<at::Tensor> & D_,
                   const c10::optional<at::Tensor> & z_,
                   const c10::optional<at::Tensor> & delta_bias_,
                   const at::Tensor & dout,
                   const c10::optional<at::Tensor> & x_,
                   const c10::optional<at::Tensor> & out_,
                   c10::optional<at::Tensor> & dz_,
                   bool delta_softplus,
                   bool recompute_out_z,
                   int height,
                   int width)
{
    auto input_type = u.scalar_type();
    auto weight_type = A.scalar_type();
    auto output_type = dout.scalar_type();

    TORCH_CHECK(input_type == at::ScalarType::Float || input_type == at::ScalarType::Half);
    TORCH_CHECK(weight_type == at::ScalarType::Float);

    const bool is_variable_B = B.dim() >= 3;
    const bool is_variable_C = C.dim() >= 3;
    const bool is_complex = weight_type == at::ScalarType::ComplexFloat;

    TORCH_CHECK(delta.scalar_type() == input_type);
    TORCH_CHECK(B.scalar_type() == (!is_variable_B ? weight_type : input_type));
    TORCH_CHECK(C.scalar_type() == (!is_variable_C ? weight_type : input_type));
    // TORCH_CHECK(dout.scalar_type() == input_type);

    TORCH_CHECK(u.is_cuda());
    TORCH_CHECK(delta.is_cuda());
    TORCH_CHECK(A.is_cuda());
    TORCH_CHECK(B.is_cuda());
    TORCH_CHECK(C.is_cuda());
    TORCH_CHECK(dout.is_cuda());

    TORCH_CHECK(u.stride(-1) == 1 || u.size(-1) == 1);
    TORCH_CHECK(delta.stride(-1) == 1 || delta.size(-1) == 1);
    TORCH_CHECK(dout.stride(-1) == 1 || dout.size(-1) == 1);

    const auto sizes = u.sizes();
    const int batch_size = sizes[0];
    const int dim = sizes[1];
    const int seqlen = sizes[2];
    const int dstate = A.size(1);
    const int n_groups = is_variable_B ? B.size(1) : 1;

    TORCH_CHECK(seqlen == height * width, "flattened input last dimention size mismatch with specified width and height");
    TORCH_CHECK(dstate <= 256, "selective_scan only supports state dimension <= 256");

    CHECK_SHAPE(u, batch_size, dim, seqlen);
    CHECK_SHAPE(delta, batch_size, dim, seqlen);
    CHECK_SHAPE(A, dim, dstate);

    if (!is_variable_B)
    {
        CHECK_SHAPE(B, dim, dstate);
    }
    else
    {
        CHECK_SHAPE(B, batch_size, n_groups, dstate, !is_complex ? seqlen : seqlen * 2);
        TORCH_CHECK(B.stride(-1) == 1 || B.size(-1) == 1);
    }

    if (!is_variable_C)
    {
        CHECK_SHAPE(C, dim, dstate);
    }
    else
    {
        CHECK_SHAPE(C, batch_size, n_groups, dstate, !is_complex ? seqlen : seqlen * 2);
        TORCH_CHECK(C.stride(-1) == 1 || C.size(-1) == 1);
    }

    CHECK_SHAPE(dout, batch_size, dim, seqlen);

    if (D_.has_value())
    {
        auto D = D_.value();
        TORCH_CHECK(D.scalar_type() == at::ScalarType::Float);
        TORCH_CHECK(D.is_cuda());
        TORCH_CHECK(D.stride(-1) == 1 || D.size(-1) == 1);
        CHECK_SHAPE(D, dim);
    }

    if (delta_bias_.has_value())
    {
        auto delta_bias = delta_bias_.value();
        TORCH_CHECK(delta_bias.scalar_type() == at::ScalarType::Float);
        TORCH_CHECK(delta_bias.is_cuda());
        TORCH_CHECK(delta_bias.stride(-1) == 1 || delta_bias.size(-1) == 1);
        CHECK_SHAPE(delta_bias, dim);
    }

    at::Tensor z, out, dz, out_z;

    const bool has_z = z_.has_value();

    if (has_z)
    {
        z = z_.value();
        TORCH_CHECK(z.scalar_type() == input_type);
        TORCH_CHECK(z.is_cuda());
        TORCH_CHECK(z.stride(-1) == 1 || z.size(-1) == 1);
        CHECK_SHAPE(z, batch_size, dim, seqlen);

        TORCH_CHECK(out_.has_value());
        out = out_.value();
        TORCH_CHECK(out.scalar_type() == output_type);
        TORCH_CHECK(out.is_cuda());
        TORCH_CHECK(out.stride(-1) == 1 || out.size(-1) == 1);
        CHECK_SHAPE(out, batch_size, dim, seqlen);

        if (dz_.has_value())
        {
            dz = dz_.value();
            TORCH_CHECK(dz.scalar_type() == input_type);
            TORCH_CHECK(dz.is_cuda());
            TORCH_CHECK(dz.stride(-1) == 1 || dz.size(-1) == 1);
            CHECK_SHAPE(dz, batch_size, dim, seqlen);
        }
        else
        {
            dz = torch::empty_like(z);
        }

        if (recompute_out_z)
        {
            out_z = torch::empty_like(out);
        }
    }

    // const int n_chunks = (seqlen + 2048 - 1) / 2048;
    const int numChunksDimX = (width + ndmamba::kMaxDimPerBlock - 1) / ndmamba::kMaxDimPerBlock;
    const int numChunksDimY = (height + ndmamba::kMaxDimPerBlock - 1) / ndmamba::kMaxDimPerBlock;

    if (1 < numChunksDimX || 1 < numChunksDimY)
    {
        TORCH_CHECK(x_.has_value());
    }

    if (x_.has_value())
    {
        auto x = x_.value();
        TORCH_CHECK(x.scalar_type() == weight_type);
        TORCH_CHECK(x.is_cuda());
        TORCH_CHECK(x.is_contiguous());

        // See L261 of selective_scan_fwd.cu
        CHECK_SHAPE(
                x,
                batch_size,
                dim,
                numChunksDimY,
                numChunksDimX,
                dstate,
                2,  // horizontal, vertical
                ndmamba::kMaxDimPerBlock,
                2   // A, Bx
        );
    }

    at::Tensor delta_a_exp_shifts = torch::empty(
            {
                    batch_size,
                    dim,
                    numChunksDimY,
                    numChunksDimX,
                    dstate,
                    2,  // horizontal, vertical
                    ndmamba::kMaxDimPerBlock,
            },
//            2333.6666f,
            u.options().dtype(weight_type)
    );

    at::Tensor du = torch::empty_like(u);
    at::Tensor ddelta = torch::empty_like(delta);
    at::Tensor dA = torch::zeros_like(A);
    at::Tensor dB = !is_variable_B ? torch::zeros_like(B) : torch::zeros_like(B, B.options().dtype(torch::kFloat32));
    at::Tensor dC = !is_variable_C ? torch::zeros_like(C) : torch::zeros_like(C, C.options().dtype(torch::kFloat32));
    at::Tensor dD;

    if (D_.has_value())
    {
        dD = torch::zeros_like(D_.value());
    }

    at::Tensor ddelta_bias;

    if (delta_bias_.has_value())
    {
        ddelta_bias = torch::zeros_like(delta_bias_.value());
    }

    SSMParamsBwd params;
    set_ssm_params_bwd(params, batch_size, dim, height, width, dstate, n_groups, ndmamba::kMaxDimPerBlock, is_variable_B, is_variable_C,
                       u, delta, A, B, C, z, out, out_z,
                       D_.has_value() ? D_.value().data_ptr() : nullptr,
                       delta_bias_.has_value() ? delta_bias_.value().data_ptr() : nullptr,
                       x_.has_value() ? x_.value().data_ptr() : nullptr,
                       dout, du, ddelta, dA, dB, dC, dz,
                       D_.has_value() ? dD.data_ptr() : nullptr,
                       delta_bias_.has_value() ? ddelta_bias.data_ptr() : nullptr,
                       has_z, delta_softplus, recompute_out_z,
                       delta_a_exp_shifts.data_ptr());

    // Otherwise the kernel will be launched from cuda:0 device
    // Cast to char to avoid compiler warning about narrowing
    at::cuda::CUDAGuard device_guard {(char) u.get_device()};
    auto stream = at::cuda::getCurrentCUDAStream().stream();

    using weight_t = float;

    DISPATCH_ITYPE_FLOAT_AND_HALF(u.scalar_type(), "selective_scan_bwd", [&]
    {
        if (output_type == input_type)
        {
            selective_scan_bwd_cuda<input_t, weight_t, input_t>(params, stream);
        }
        else
        {
            selective_scan_bwd_cuda<input_t, weight_t, float>(params, stream);
        }
    });

    std::vector<at::Tensor> result = {du, ddelta, dA, dB.to(B.dtype()), dC.to(C.dtype()), dD, ddelta_bias};

    if (has_z)
    {
        result.push_back(dz);
    }

    if (recompute_out_z)
    {
        result.push_back(out_z);
    }

    return result;
}